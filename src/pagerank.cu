#include <algorithm>
#include <chrono>
#include <hip/hip_runtime.h>
#include <fstream>
#include <iostream>
#include <sstream>
#include <string>
#include <unordered_map>
#include <vector>

static constexpr double DAMPING_FACTOR = 0.85;

__global__ void pagerank_kernel(int num_nodes, int *row_offsets,
                                int *col_indices, int *out_degrees,
                                double *rank_old, double *rank_new,
                                double dangling_sum) {
  int i = blockDim.x * blockIdx.x + threadIdx.x;
  if (i < num_nodes) {
    double sum = dangling_sum / num_nodes;
    int row_start = row_offsets[i];
    int row_end = row_offsets[i + 1];
    for (int idx = row_start; idx < row_end; ++idx) {
      int j = col_indices[idx]; // Node j links to node i
      int out_degree = out_degrees[j];
      if (out_degree > 0) {
        sum += rank_old[j] / out_degree;
      }
    }
    rank_new[i] = (1.0 - DAMPING_FACTOR) / num_nodes + DAMPING_FACTOR * sum;
  }
}

__global__ void compute_dangling_sum(int num_nodes, int *out_degrees,
                                     double *rank_old,
                                     double *dangling_contrib) {
  __shared__ double sdata[256];
  int tid = threadIdx.x;
  int i = blockIdx.x * blockDim.x + tid;
  double my_sum = 0.0;
  if (i < num_nodes && out_degrees[i] == 0) {
    my_sum = rank_old[i];
  }
  sdata[tid] = my_sum;
  __syncthreads();

  for (unsigned int s = blockDim.x / 2; s > 0; s >>= 1) {
    if (tid < s) {
      sdata[tid] += sdata[tid + s];
    }
    __syncthreads();
  }

  if (tid == 0) {
    atomicAdd(dangling_contrib, sdata[0]);
  }
}

void build_graph(const std::string &filepath, int &num_nodes, int &num_edges,
                 std::vector<int> &row_offsets, std::vector<int> &col_indices,
                 std::vector<int> &out_degrees, std::vector<int> &node_ids) {
  std::ifstream file(filepath);
  if (!file.is_open()) {
    throw std::runtime_error("Could not open file");
  }
  std::string line;
  std::unordered_map<int, int> node_id_to_index;
  num_nodes = 0;
  num_edges = 0;

  std::vector<std::vector<int>> adj_incoming;
  std::vector<int> out_degrees_temp;

  while (std::getline(file, line)) {
    std::stringstream ss(line);
    std::string token;
    std::getline(ss, token, ',');
    int from = std::stoi(token);
    std::getline(ss, token, ',');
    int to = std::stoi(token);

    if (node_id_to_index.find(from) == node_id_to_index.end()) {
      node_id_to_index[from] = num_nodes++;
      node_ids.push_back(from);
      adj_incoming.push_back(std::vector<int>());
      out_degrees_temp.push_back(0);
    }
    if (node_id_to_index.find(to) == node_id_to_index.end()) {
      node_id_to_index[to] = num_nodes++;
      node_ids.push_back(to);
      adj_incoming.push_back(std::vector<int>());
      out_degrees_temp.push_back(0);
    }

    int from_idx = node_id_to_index[from];
    int to_idx = node_id_to_index[to];

    adj_incoming[to_idx].push_back(from_idx);
    num_edges++;

    out_degrees_temp[from_idx]++;
  }

  row_offsets.resize(num_nodes + 1);
  row_offsets[0] = 0;
  for (int i = 0; i < num_nodes; ++i) {
    row_offsets[i + 1] = row_offsets[i] + adj_incoming[i].size();
  }

  col_indices.resize(num_edges);
  int idx = 0;
  for (int i = 0; i < num_nodes; ++i) {
    for (int j = 0; j < adj_incoming[i].size(); ++j) {
      col_indices[idx++] = adj_incoming[i][j];
    }
  }

  out_degrees = out_degrees_temp;
}

void write_rank(const std::vector<int> &node_ids,
                const std::vector<double> &rank, const std::string &filepath) {
  std::ofstream file(filepath);
  if (!file.is_open()) {
    throw std::runtime_error("Could not open file");
  }
  std::vector<int> indices(rank.size());
  for (int i = 0; i < rank.size(); ++i) {
    indices[i] = i;
  }
  std::sort(indices.begin(), indices.end(),
            [&rank](int a, int b) { return rank[a] > rank[b]; });
  for (const auto &idx : indices) {
    file << node_ids[idx] << "," << rank[idx] << std::endl;
  }
}

int main(int argc, char *argv[]) {
  auto start = std::chrono::high_resolution_clock::now();
  std::string input_file = std::string(argv[1]);
  std::string output_file = std::string(argv[2]);

  int num_nodes, num_edges;
  std::vector<int> row_offsets, col_indices, out_degrees, node_ids;
  build_graph(input_file, num_nodes, num_edges, row_offsets, col_indices,
              out_degrees, node_ids);

  int *d_row_offsets, *d_col_indices, *d_out_degrees;
  double *d_rank_old, *d_rank_new, *d_dangling_contrib;
  hipMalloc(&d_row_offsets, (num_nodes + 1) * sizeof(int));
  hipMalloc(&d_col_indices, num_edges * sizeof(int));
  hipMalloc(&d_out_degrees, num_nodes * sizeof(int));
  hipMalloc(&d_rank_old, num_nodes * sizeof(double));
  hipMalloc(&d_rank_new, num_nodes * sizeof(double));
  hipMalloc(&d_dangling_contrib, sizeof(double));

  hipMemcpy(d_row_offsets, row_offsets.data(), (num_nodes + 1) * sizeof(int),
             hipMemcpyHostToDevice);
  hipMemcpy(d_col_indices, col_indices.data(), num_edges * sizeof(int),
             hipMemcpyHostToDevice);
  hipMemcpy(d_out_degrees, out_degrees.data(), num_nodes * sizeof(int),
             hipMemcpyHostToDevice);

  double initial_rank = 1.0 / num_nodes;
  std::vector<double> rank_old(num_nodes, initial_rank);
  hipMemcpy(d_rank_old, rank_old.data(), num_nodes * sizeof(double),
             hipMemcpyHostToDevice);

  int max_iters = 100;
  int block_size = 256;
  int grid_size = (num_nodes + block_size - 1) / block_size;
  auto preamble_time = std::chrono::high_resolution_clock::now();
  auto preamble_time_seconds =
      std::chrono::duration_cast<std::chrono::milliseconds>(preamble_time -
                                                            start)
          .count();
  for (int iter = 0; iter < max_iters; ++iter) {
    double zero = 0.0;
    hipMemcpy(d_dangling_contrib, &zero, sizeof(double),
               hipMemcpyHostToDevice);

    compute_dangling_sum<<<grid_size, block_size>>>(
        num_nodes, d_out_degrees, d_rank_old, d_dangling_contrib);
    hipDeviceSynchronize();

    double dangling_contrib;
    hipMemcpy(&dangling_contrib, d_dangling_contrib, sizeof(double),
               hipMemcpyDeviceToHost);

    pagerank_kernel<<<grid_size, block_size>>>(
        num_nodes, d_row_offsets, d_col_indices, d_out_degrees, d_rank_old,
        d_rank_new, dangling_contrib);
    hipDeviceSynchronize();

    std::swap(d_rank_old, d_rank_new);
  }
  auto main_loop_time = std::chrono::high_resolution_clock::now();
  auto main_loop_time_seconds =
      std::chrono::duration_cast<std::chrono::milliseconds>(main_loop_time -
                                                            preamble_time)
          .count();
  hipMemcpy(rank_old.data(), d_rank_old, num_nodes * sizeof(double),
             hipMemcpyDeviceToHost);

  write_rank(node_ids, rank_old, output_file);

  hipFree(d_row_offsets);
  hipFree(d_col_indices);
  hipFree(d_out_degrees);
  hipFree(d_rank_old);
  hipFree(d_rank_new);
  hipFree(d_dangling_contrib);
  auto end = std::chrono::high_resolution_clock::now();
  auto total_time_seconds =
      std::chrono::duration_cast<std::chrono::milliseconds>(end - start)
          .count();
  auto write_time_seconds =
      total_time_seconds - preamble_time_seconds - main_loop_time_seconds;
  std::cout << "Preamble: " << preamble_time_seconds << " ms" << std::endl;
  std::cout << "Main Algorithm: " << main_loop_time_seconds << " ms"
            << std::endl;
  std::cout << "Write Output: " << write_time_seconds << " ms" << std::endl;
  std::cout << "Total: " << total_time_seconds << " ms" << std::endl;
  return 0;
}
