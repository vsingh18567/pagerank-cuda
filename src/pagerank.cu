#include <algorithm>
#include <hip/hip_runtime.h>
#include <fstream>
#include <iostream>
#include <sstream>
#include <string>
#include <unordered_map>
#include <vector>

static constexpr double DAMPING_FACTOR = 0.85;

__global__ void pagerank_kernel(int num_nodes, int *row_offsets,
                                int *col_indices, int *out_degrees,
                                double *rank_old, double *rank_new,
                                double dangling_sum) {
  int i = blockDim.x * blockIdx.x + threadIdx.x;
  if (i < num_nodes) {
    double sum = dangling_sum / num_nodes;
    int row_start = row_offsets[i];
    int row_end = row_offsets[i + 1];
    for (int idx = row_start; idx < row_end; ++idx) {
      int j = col_indices[idx]; // Node j links to node i
      int out_degree = out_degrees[j];
      if (out_degree > 0) {
        sum += rank_old[j] / out_degree;
      }
    }
    rank_new[i] = (1.0 - DAMPING_FACTOR) / num_nodes + DAMPING_FACTOR * sum;
  }
}

__global__ void compute_dangling_sum(int num_nodes, int *out_degrees,
                                     double *rank_old,
                                     double *dangling_contrib) {
  __shared__ double sdata[256];
  int tid = threadIdx.x;
  int i = blockIdx.x * blockDim.x + tid;
  double my_sum = 0.0;
  if (i < num_nodes && out_degrees[i] == 0) {
    my_sum = rank_old[i];
  }
  sdata[tid] = my_sum;
  __syncthreads();

  // Reduction in shared memory
  for (unsigned int s = blockDim.x / 2; s > 0; s >>= 1) {
    if (tid < s) {
      sdata[tid] += sdata[tid + s];
    }
    __syncthreads();
  }

  // Write result from each block to global memory
  if (tid == 0) {
    atomicAdd(dangling_contrib, sdata[0]);
  }
}

void build_graph(const std::string &filepath, int &num_nodes, int &num_edges,
                 std::vector<int> &row_offsets, std::vector<int> &col_indices,
                 std::vector<int> &out_degrees, std::vector<int> &node_ids) {
  std::ifstream file(filepath);
  if (!file.is_open()) {
    throw std::runtime_error("Could not open file");
  }
  std::string line;
  std::unordered_map<int, int> node_id_to_index;
  num_nodes = 0;
  num_edges = 0;

  std::vector<std::vector<int>> adj_incoming;
  std::vector<int> out_degrees_temp;

  while (std::getline(file, line)) {
    std::stringstream ss(line);
    std::string token;
    std::getline(ss, token, ',');
    int from = std::stoi(token);
    std::getline(ss, token, ',');
    int to = std::stoi(token);

    // Map node IDs to indices
    if (node_id_to_index.find(from) == node_id_to_index.end()) {
      node_id_to_index[from] = num_nodes++;
      node_ids.push_back(from);
      adj_incoming.push_back(std::vector<int>());
      out_degrees_temp.push_back(0);
    }
    if (node_id_to_index.find(to) == node_id_to_index.end()) {
      node_id_to_index[to] = num_nodes++;
      node_ids.push_back(to);
      adj_incoming.push_back(std::vector<int>());
      out_degrees_temp.push_back(0);
    }

    int from_idx = node_id_to_index[from];
    int to_idx = node_id_to_index[to];

    // Build adjacency list of incoming edges
    adj_incoming[to_idx].push_back(from_idx);
    num_edges++;

    // Increment out-degree of 'from' node
    out_degrees_temp[from_idx]++;
  }

  // Build CSR arrays
  row_offsets.resize(num_nodes + 1);
  row_offsets[0] = 0;
  for (int i = 0; i < num_nodes; ++i) {
    row_offsets[i + 1] = row_offsets[i] + adj_incoming[i].size();
  }

  col_indices.resize(num_edges);
  int idx = 0;
  for (int i = 0; i < num_nodes; ++i) {
    for (int j = 0; j < adj_incoming[i].size(); ++j) {
      col_indices[idx++] = adj_incoming[i][j];
    }
  }

  out_degrees = out_degrees_temp;
}

void write_rank(const std::vector<int> &node_ids,
                const std::vector<double> &rank, const std::string &filepath) {
  std::ofstream file(filepath);
  if (!file.is_open()) {
    throw std::runtime_error("Could not open file");
  }
  // Create a vector of indices
  std::vector<int> indices(rank.size());
  for (int i = 0; i < rank.size(); ++i) {
    indices[i] = i;
  }
  // Sort indices based on rank
  std::sort(indices.begin(), indices.end(),
            [&rank](int a, int b) { return rank[a] > rank[b]; });
  for (const auto &idx : indices) {
    file << node_ids[idx] << "," << rank[idx] << std::endl;
  }
}

int main(int argc, char *argv[]) {
  std::string input_file = std::string(argv[1]);
  std::string output_file = std::string(argv[2]);

  int num_nodes, num_edges;
  std::vector<int> row_offsets, col_indices, out_degrees, node_ids;
  build_graph(input_file, num_nodes, num_edges, row_offsets, col_indices,
              out_degrees, node_ids);

  // Allocate device memory
  int *d_row_offsets, *d_col_indices, *d_out_degrees;
  double *d_rank_old, *d_rank_new, *d_dangling_contrib;
  hipMalloc(&d_row_offsets, (num_nodes + 1) * sizeof(int));
  hipMalloc(&d_col_indices, num_edges * sizeof(int));
  hipMalloc(&d_out_degrees, num_nodes * sizeof(int));
  hipMalloc(&d_rank_old, num_nodes * sizeof(double));
  hipMalloc(&d_rank_new, num_nodes * sizeof(double));
  hipMalloc(&d_dangling_contrib, sizeof(double));

  // Copy data to device
  hipMemcpy(d_row_offsets, row_offsets.data(), (num_nodes + 1) * sizeof(int),
             hipMemcpyHostToDevice);
  hipMemcpy(d_col_indices, col_indices.data(), num_edges * sizeof(int),
             hipMemcpyHostToDevice);
  hipMemcpy(d_out_degrees, out_degrees.data(), num_nodes * sizeof(int),
             hipMemcpyHostToDevice);

  // Initialize rank_old
  double initial_rank = 1.0 / num_nodes;
  std::vector<double> rank_old(num_nodes, initial_rank);
  hipMemcpy(d_rank_old, rank_old.data(), num_nodes * sizeof(double),
             hipMemcpyHostToDevice);

  // PageRank iterations
  int max_iters = 100;
  int block_size = 256;
  int grid_size = (num_nodes + block_size - 1) / block_size;

  for (int iter = 0; iter < max_iters; ++iter) {
    // Reset dangling_contrib to zero
    double zero = 0.0;
    hipMemcpy(d_dangling_contrib, &zero, sizeof(double),
               hipMemcpyHostToDevice);

    // Compute dangling sum
    compute_dangling_sum<<<grid_size, block_size>>>(
        num_nodes, d_out_degrees, d_rank_old, d_dangling_contrib);
    hipDeviceSynchronize();

    // Copy dangling_contrib back to host
    double dangling_contrib;
    hipMemcpy(&dangling_contrib, d_dangling_contrib, sizeof(double),
               hipMemcpyDeviceToHost);

    // PageRank kernel
    pagerank_kernel<<<grid_size, block_size>>>(
        num_nodes, d_row_offsets, d_col_indices, d_out_degrees, d_rank_old,
        d_rank_new, dangling_contrib);
    hipDeviceSynchronize();

    // Swap rank_old and rank_new
    std::swap(d_rank_old, d_rank_new);
  }

  // Copy ranks back to host
  hipMemcpy(rank_old.data(), d_rank_old, num_nodes * sizeof(double),
             hipMemcpyDeviceToHost);

  // Write ranks to output file
  write_rank(node_ids, rank_old, output_file);

  // Free device memory
  hipFree(d_row_offsets);
  hipFree(d_col_indices);
  hipFree(d_out_degrees);
  hipFree(d_rank_old);
  hipFree(d_rank_new);
  hipFree(d_dangling_contrib);

  return 0;
}
